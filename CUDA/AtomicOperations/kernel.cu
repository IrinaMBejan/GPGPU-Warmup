#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>

#define NUM_THREADS 1000000
#define ARRAY_SIZE  100

#define BLOCK_WIDTH 1000

__global__ void increment_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;
	atomicAdd(&g[i], 1);
}


void print_array(int *array, int size)
{
	printf("{ ");
	for (int i = 0; i < size; i++) { printf("%d ", array[i]); }
	printf("}\n");
}

int main(int argc, char **argv)
{
	printf("%d total threads in %d blocks writing into %d array elements\n",
		NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

	int h_array[ARRAY_SIZE];
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	int * d_array;
	hipMalloc((void **)&d_array, ARRAY_BYTES);
	hipMemset((void *)d_array, 0, ARRAY_BYTES);

	increment_atomic << <NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >> >(d_array);

	hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
	print_array(h_array, ARRAY_SIZE);

	hipFree(d_array);
	return 0;
}